#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <limits>
#include "vanilla_impl.cuh"

__global__ void SelectionPolicy(int *explore_count, float *total_reward, int *parent_explore_count, float *uct_ptr,
                                float *uct_value) {
  if (*explore_count == 0) {
    uct_value[0] = std::numeric_limits<float>::infinity();
    return;
  }
  uct_value[0] =
    *total_reward / *explore_count + *uct_ptr * std::sqrt(std::log(*parent_explore_count) / *explore_count);
  return;
}

__global__ void Update(int *explore_count, float *total_reward, float *values, int player) {
  *explore_count += 1;
  *total_reward += values[player];
}

void CalSelectionPolicy(int *explore_count, float *total_reward, int *parent_explore_count, float *uct_ptr,
                        float *uct_value, hipStream_t hip_stream) {
  dim3 blockSize(1);
  dim3 gridSize(1);
  SelectionPolicy<<<gridSize, blockSize, 0, hip_stream>>>(explore_count, total_reward, parent_explore_count, uct_ptr,
                                                           uct_value);
  return;
}

void CalUpdate(int *explore_count, float *total_reward, float *values, int player, hipStream_t hip_stream) {
  dim3 blockSize(1);
  dim3 gridSize(1);
  Update<<<gridSize, blockSize, 0, hip_stream>>>(explore_count, total_reward, values, player);
  return;
}
